#include "cuda.cuh"

#include <glog/logging.h>

using namespace std;
using namespace glinthawk::models::common::cuda;

void CHECK_CUBLAS( const hipblasStatus_t err, const source_location location )
{
  if ( err != HIPBLAS_STATUS_SUCCESS ) {
    LOG( FATAL ) << "CUBLAS error "s << cublasGetStatusName( err ) << ": " << cublasGetStatusString( err ) << " ("
                 << location.file_name() << ":" << to_string( location.line() ) << ")";
  }
}

void CHECK_CUDA( const hipError_t err, const source_location location )
{
  if ( err != hipSuccess ) {
    LOG( FATAL ) << "CUDA error " << string( hipGetErrorName( err ) ) << ": " << string( hipGetErrorString( err ) )
                 << " (" << location.file_name() << ":" << to_string( location.line() ) << ")";
  }
}
