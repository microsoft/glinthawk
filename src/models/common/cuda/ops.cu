#include "hip/hip_runtime.h"
#include "ops.cuh"

#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include "util/exception.hh"

using namespace std;

namespace glinthawk::models::common::cuda::ops {

namespace {
hipblasHandle_t cublas_handle;
}

void init() { hipblasCreate( &cublas_handle ); }

template<typename DType>
__global__ void normalize_and_scale( DType* output,
                                     const DType* x,
                                     const DType* weight,
                                     const int size,
                                     const float* ss )
{
  const int i = threadIdx.x;
  output[i] = weight[i] * __float2half( ( *ss ) * __half2float( x[i] ) );
}

template<>
void rmsnorm<float>( float* output, const float* x, const float* weight, const int size )
{
  // calculate sum of squares
  float ss = 0.0f;

  hipblasSdot( cublas_handle, size, x, 1, x, 1, &ss );
  ss /= size;
  ss += 1e-5f;
  ss = 1.0f / sqrtf( ss );

  // normalize_and_scale<<<1, size>>>( output, x, weight, size, ss );
}

__global__ void print_this( const __half* x, const int size, float* output )
{
  float result = 0.0;
  for ( int i = 0; i < size; i++ ) {
    float x_f = __half2float( x[i] );
    result += x_f * x_f;
  }

  *output = result;

  *output /= size;
  *output += 1e-5f;
  *output = 1.0f / sqrtf( *output );
}

template<>
void rmsnorm<__half>( __half* output, const __half* x, const __half* weight, const int size )
{
  // calculate sum of squares
  float* ss;
  hipMalloc( &ss, sizeof( float ) );

  // hipblasDotEx( cublas_handle, size, x, HIP_R_16F, 1, x, HIP_R_16F, 1, &ss, HIP_R_32F, HIP_R_32F );
  print_this<<<1, 1>>>( x, size, ss );
  normalize_and_scale<<<1, size>>>( output, x, weight, size, ss );

  hipFree( ss );
}

template<>
void softmax<float>( float* _x, const int size )
{
  thrust::device_ptr<float> x { _x };

  const float max_val = *thrust::max_element( x, x + size );
  const float sum = thrust::transform_reduce(
    x, x + size, [max_val] __device__( const float x ) { return expf( x - max_val ); }, 0.0f, thrust::plus<float>() );
  thrust::transform( x, x + size, x, [sum] __device__( const float x ) { return x / sum; } );
}

template<>
void softmax( __half* _x, const int size )
{
  thrust::device_ptr<__half> x { _x };

  const __half max_val = *thrust::max_element( x, x + size );
  const __half sum = thrust::transform_reduce(
    x,
    x + size,
    [max_val] __device__( const __half x ) { return hexp( x - max_val ); },
    __half(),
    thrust::plus<__half>() );
  thrust::transform( x, x + size, x, [sum] __device__( const __half x ) { return x / sum; } );
}

template<typename DType>
void sample( const DType* probabilities, const int n, int* output )
{
  throw runtime_error( "not implemented" );
}

template<typename DType>
void argmax( const DType* _v, const int n, int* _output )
{
  thrust::device_ptr<const DType> v { _v };
  thrust::device_ptr<int> output { _output };

  const auto it = thrust::max_element( v, v + n );
  *output = thrust::distance( v, it );
}

template<>
void accum<float>( float* a, const float* b, const int size )
{
  float alpha = 1.0f;
  hipblasSaxpy( cublas_handle, size, &alpha, b, 1, a, 1 );
}

__global__ void accum_this( __half* a, const __half* b, const int size )
{
  for ( int i = 0; i < size; i++ ) {
    a[i] = a[i] + b[i];
  }
}

template<>
void accum<__half>( __half* a, const __half* b, const int size )
{
  __half alpha = 1.0f;
  // hipblasAxpyEx( cublas_handle, size, &alpha, HIP_R_16F, b, HIP_R_16F, 1, a, HIP_R_16F, 1, HIP_R_16F );
  accum_this<<<1, 1>>>( a, b, size );
}

// void rmsnorm( float* o, const float* x, const float* weight, const int size );
// void softmax( float* x, const int size );

template<>
void matmul<float>( float* xout, const float* x, const float* W, const int n, const int d )
{
  float alpha = 1.0f;
  float beta = 0.0f;

  // W(d,n) @ x(n,) -> xout(d,)
  hipblasSgemv( cublas_handle, HIPBLAS_OP_T, n, d, &alpha, W, n, x, 1, &beta, xout, 1 );
}

template<>
void matmul<__half>( __half* xout, const __half* x, const __half* W, const int s, const int r )
{
  __half alpha = 1.0f;
  __half beta = 0.0f;

  // W(r,s) @ x(s,) -> xout(r,)
  const int m = 1;
  const int n = r;
  const int k = s;
  const int lda = m;
  const int ldb = k;
  const int ldc = m;
  hipblasHgemm( cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, x, lda, W, ldb, &beta, xout, ldc );
}

template<>
void silu<float>( float* _hb, float* _hb2, const int hidden_dim )
{
  thrust::device_ptr<float> hb { _hb };
  thrust::device_ptr<float> hb2 { _hb2 };

  thrust::transform( hb, hb + hidden_dim, hb, [] __device__( float x ) { return ( x / ( 1.0f + expf( -x ) ) ); } );
  thrust::transform( hb, hb + hidden_dim, hb2, hb, thrust::multiplies<float>() );
}

// __global__ void silu_half( __half* hb, __half* hb2, const int hidden_dim )
// {
//   const int t = threadIdx.x;
//   const int b = blockIdx.x;
//   const int i = b * blockDim.x + t;

//   if ( i < hidden_dim ) {
//     hb[i] = hb[i] * ( static_cast<__half>( 1.0f ) / ( static_cast<__half>( 1.0f ) + hexp( -hb[i] ) ) );
//     hb[i] = hb[i] * hb2[i];
//   }
// }

template<>
void silu<__half>( __half* _hb, __half* _hb2, const int hidden_dim )
{
  thrust::device_ptr<__half> hb { _hb };
  thrust::device_ptr<__half> hb2 { _hb2 };

  thrust::transform( hb, hb + hidden_dim, hb, [] __device__( __half x ) {
    return ( x / ( static_cast<__half>( 1.0f ) + hexp( -x ) ) );
  } );

  thrust::transform( hb, hb + hidden_dim, hb2, hb, thrust::multiplies<__half>() );
}

template void rmsnorm<float>( float* output, const float* x, const float* weight, const int size );
template void rmsnorm<__half>( __half* output, const __half* x, const __half* weight, const int size );

template void argmax<float>( const float* v, const int n, int* output );
template void argmax<__half>( const __half* v, const int n, int* output );

template void sample<float>( const float* probabilities, const int n, int* output );
template void sample<__half>( const __half* probabilities, const int n, int* output );

template void accum<float>( float* a, const float* b, const int size );
template void accum<__half>( __half* a, const __half* b, const int size );

template void softmax<float>( float* x, const int size );
template void softmax<__half>( __half* x, const int size );

template void matmul<float>( float* xout, const float* x, const float* w, const int n, const int d );
template void matmul<__half>( __half* xout, const __half* x, const __half* w, const int n, const int d );

} // namespace glinthawk::models::common::cuda
