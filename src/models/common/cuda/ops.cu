#include "hip/hip_runtime.h"
#include "ops.cuh"

#include <source_location>
#include <string>

#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include "util/exception.hh"

using namespace std;

namespace glinthawk::models::common::cuda::ops {

namespace {
hipblasHandle_t cublas_handle;
}

void CHECK_CUBLAS( const hipblasStatus_t err, const source_location location = source_location::current() )
{
  if ( err != HIPBLAS_STATUS_SUCCESS ) {
    throw runtime_error( "CUBLAS error "s + cublasGetStatusName( err ) + ": " + cublasGetStatusString( err ) + " ("
                         + location.file_name() + ":" + std::to_string( location.line() ) + ")" );
  }
}

void init() { hipblasCreate( &cublas_handle ); }
void destroy() { hipblasDestroy( cublas_handle ); }

__global__ void normalize_and_scale_full( float* output,
                                          const float* x,
                                          const float* weight,
                                          const int size,
                                          const float ss )
{
  const int i = threadIdx.x + blockIdx.x * 64;
  if (i < size)
    output[i] = weight[i] * ss * x[i];
}

__global__ void normalize_and_scale_half( __half* output,
                                          const __half* x,
                                          const __half* weight,
                                          const int size,
                                          const float ss )
{
  const int i = threadIdx.x + blockIdx.x * 64;
  if (i < size)
    output[i] = weight[i] * __float2half( ss * __half2float( x[i] ) );
}

template<>
void rmsnorm<float>( float* output, const float* x, const float* weight, const int size )
{
  // calculate sum of squares
  float ss = 0.0f;

  CHECK_CUBLAS( hipblasSdot( cublas_handle, size, x, 1, x, 1, &ss ) );
  ss /= size;
  ss += 1e-5f;
  ss = 1.0f / sqrtf( ss );

  normalize_and_scale_full<<<(size+63)/64, 1024>>>( output, x, weight, size, ss );
}

struct square : public thrust::unary_function<__half,float>
{
  __host__ __device__
    float operator()(const __half& x) const {
      const float x_f = __half2float(x);
      return x_f * x_f;
  }
};

template<>
void rmsnorm<__half>( __half* output, const __half* x, const __half* weight, const int size )
{
  // calculate sum of squares
  thrust::device_ptr<__half> thrust_x { const_cast<__half*>(x) };
  float ss = thrust::transform_reduce(thrust_x, thrust_x + size, square(), 0.0f, thrust::plus<float>() );
  ss /= size;
  ss += 1e-5f;
  ss = 1.0f / sqrtf( ss );

  normalize_and_scale_half<<<(size+63)/64, 64>>>( output, x, weight, size, ss );
}

template<>
void softmax<float>( float* _x, const int size )
{
  thrust::device_ptr<float> x { _x };

  const float max_val = *thrust::max_element( x, x + size );
  const float sum = thrust::transform_reduce(
    x, x + size, [max_val] __device__( const float x ) { return expf( x - max_val ); }, 0.0f, thrust::plus<float>() );
  thrust::transform( x, x + size, x, [sum] __device__( const float x ) { return x / sum; } );
}

template<>
void softmax( __half* _x, const int size )
{
  thrust::device_ptr<__half> x { _x };
  const __half max_val = *thrust::max_element( x, x + size );
  const __half sum = thrust::transform_reduce(
    x,
    x + size,
    [max_val] __device__( const __half x ) { return hexp( x - max_val ); },
    __half(),
    thrust::plus<__half>() );
  thrust::transform( x, x + size, x, [sum] __device__( const __half x ) { return x / sum; } );
}

template<typename DType>
void attention_0_gemm(const DType* query,
                      const DType* key,
                      DType* att,
                      const int n_layers,
                      const int seq_len,
                      const int head_size,
                      const int n_heads,
                      const int n_tokens)
{
  const float alpha = 1.0f / sqrtf( head_size );
  const float beta = 0.0f;

  const int m = 1;
  const int n = n_tokens;
  const int k = head_size;

  const int lda = m;
  const int ldb = n_layers * n_heads * head_size * 2;
  const int ldc = m;

  const int strideA = head_size;
  const int strideB = head_size;
  const int strideC = seq_len;

  const int batchCount = n_heads;

  if constexpr ( is_same_v<DType, __half> ) {
        CHECK_CUBLAS(
hipblasGemmStridedBatchedEx( cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, query, HIP_R_16F, lda, strideA,
                        key, HIP_R_16F, ldb, strideB, &beta, att, HIP_R_16F, ldc, strideC, batchCount, HIP_R_32F,
                        HIPBLAS_GEMM_DEFAULT ) );
  } else {
      CHECK_CUBLAS(
  hipblasGemmStridedBatchedEx( cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, query, HIP_R_32F, lda, strideA,
                              key, HIP_R_32F, ldb, strideB, &beta, att, HIP_R_32F, ldc, strideC, batchCount, HIP_R_32F,
                              HIPBLAS_GEMM_DEFAULT ) );
  }

}

template<typename DType>
void attention_2_gemm(const DType* att,
                       const DType* value,
                       DType* xb,
                       const int n_layers,
                       const int seq_len,
                       const int head_size,
                       const int n_heads,
                       const int n_tokens)
{
  const float alpha = 1.0f;
  const float beta = 0.0f;

  const int m = head_size;
  const int n = 1;
  const int k = n_tokens;

  const int lda = n_layers * n_heads * head_size * 2;
  const int ldb = k;
  const int ldc = m;

  const int strideA = head_size;
  const int strideB = seq_len;
  const int strideC = head_size;

  const int batchCount = n_heads;

  if constexpr ( is_same_v<DType, __half> ) {
      CHECK_CUBLAS(
        hipblasGemmStridedBatchedEx( cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, value, HIP_R_16F, lda, strideA,
                                    att, HIP_R_16F, ldb, strideB, &beta, xb, HIP_R_16F, ldc, strideC, batchCount, HIP_R_32F,
                                    HIPBLAS_GEMM_DEFAULT ) );
  } else {
      CHECK_CUBLAS(
        hipblasGemmStridedBatchedEx( cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, value, HIP_R_32F, lda, strideA,
                                    att, HIP_R_32F, ldb, strideB, &beta, xb, HIP_R_32F, ldc, strideC, batchCount, HIP_R_32F,
                                    HIPBLAS_GEMM_DEFAULT ) );
  }

}

template<typename DType>
void sample( const DType* probabilities, const int n, int* output )
{
  throw runtime_error( "not implemented" );
}

template<typename DType>
void argmax( const DType* _v, const int n, int* _output )
{
  thrust::device_ptr<const DType> v { _v };
  thrust::device_ptr<int> output { _output };

  const auto it = thrust::max_element( v, v + n );
  *output = thrust::distance( v, it );
}

template<>
void accum<float>( float* a, const float* b, const int size )
{
  float alpha = 1.0f;
  hipblasSaxpy( cublas_handle, size, &alpha, b, 1, a, 1 );
}

__global__ void accum_cuda(__half* a, const __half* b, const int size){
  const int i = blockIdx.x * 64 + threadIdx.x;
  a[i] += b[i];
}

template<>
void accum<__half>( __half* a, const __half* b, const int size )
{
//  float alpha = 1.0f;
//  CHECK_CUBLAS(
//    hipblasAxpyEx( cublas_handle, size, &alpha, HIP_R_32F, b, HIP_R_16F, 1, a, HIP_R_16F, 1, HIP_R_32F ) );
  accum_cuda<<<(size+63)/64, 64>>>(a, b, size);
}

// void rmsnorm( float* o, const float* x, const float* weight, const int size );
// void softmax( float* x, const int size );

template<>
void matmul<float>( float* xout, const float* x, const float* W, const int n, const int d )
{
  float alpha = 1.0f;
  float beta = 0.0f;

  // W(d,n) @ x(n,) -> xout(d,)
  CHECK_CUBLAS( hipblasSgemv( cublas_handle, HIPBLAS_OP_T, n, d, &alpha, W, n, x, 1, &beta, xout, 1 ) );
}

template<>
void matmul<__half>( __half* xout, const __half* x, const __half* W, const int s, const int r )
{
  float alpha = 1.0f;
  float beta = 0.0f;

  // W(r,s) @ x(s,) -> xout(r,)
  const int m = 1;
  const int n = r;
  const int k = s;
  const int lda = m;
  const int ldb = k;
  const int ldc = m;

  CHECK_CUBLAS(
    hipblasGemmEx(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha, x, HIP_R_16F, lda, W, HIP_R_16F, ldb,
                  &beta, xout, HIP_R_16F, ldc, HIP_R_32F, HIPBLAS_GEMM_DEFAULT ) );
}

__global__ void silu_direct(float* _hb, const float* _hb2, const int hidden_dim) {
  const int i = threadIdx.x + blockIdx.x * 64;
  if (i < hidden_dim){
    const float x = _hb[i];
    _hb[i] = x / (1.0f + expf(-x)) * _hb2[i];
  }
}

__global__ void silu_direct(__half* _hb, const __half* _hb2, const int hidden_dim) {
  const int i = threadIdx.x + blockIdx.x * 64;
  if (i < hidden_dim){
    const __half x = _hb[i];
    _hb[i] = x / (__half(1.0f) + hexp(-x)) * _hb2[i];
  }
}

template<>
void silu<float>( float* _hb, float* _hb2, const int hidden_dim )
{
  silu_direct<<<(hidden_dim+63)/64, 64>>>( _hb, _hb2, hidden_dim );
}

template<>
void silu<__half>( __half* _hb, __half* _hb2, const int hidden_dim )
{
  silu_direct<<<(hidden_dim+63)/64, 64>>>( _hb, _hb2, hidden_dim );
}

template void matmul<float>( float* xout, const float* x, const float* w, const int n, const int d );
template void matmul<__half>( __half* xout, const __half* x, const __half* w, const int n, const int d );

template void rmsnorm<float>( float* output, const float* x, const float* weight, const int size );
template void rmsnorm<__half>( __half* output, const __half* x, const __half* weight, const int size );

template void argmax<float>( const float* v, const int n, int* output );
template void argmax<__half>( const __half* v, const int n, int* output );

template void sample<float>( const float* probabilities, const int n, int* output );
template void sample<__half>( const __half* probabilities, const int n, int* output );

template void accum<float>( float* a, const float* b, const int size );
template void accum<__half>( __half* a, const __half* b, const int size );

template void softmax<float>( float* x, const int size );
template void softmax<__half>( __half* x, const int size );

template void matmul<float>( float* xout, const float* x, const float* w, const int n, const int d );
template void matmul<__half>( __half* xout, const __half* x, const __half* w, const int n, const int d );

template void attention_0_gemm<float>(const float* query,
                       const float* key,
                                       float* att,
                       const int n_layers,
                       const int seq_len,
                       const int head_size,
                       const int n_heads,
                       const int n_tokens);
template void attention_0_gemm<__half>(const __half* query,
                                       const __half* key,
                                        __half* att,
                                       const int n_layers,
                                       const int seq_len,
                                       const int head_size,
                                       const int n_heads,
                                       const int n_tokens);

template void attention_2_gemm<float>(const float* query,
                                       const float* key,
                                       float* att,
                                       const int n_layers,
                                       const int seq_len,
                                       const int head_size,
                                       const int n_heads,
                                       const int n_tokens);
template void attention_2_gemm<__half>(const __half* query,
                                        const __half* key,
                                        __half* att,
                                        const int n_layers,
                                        const int seq_len,
                                        const int head_size,
                                        const int n_heads,
                                        const int n_tokens);

} // namespace glinthawk::models::common::cuda
