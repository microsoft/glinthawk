#include "cudainfo.cuh"

#include <iostream>

#include <hip/hip_runtime.h>
#include <glog/logging.h>

using namespace std;
using namespace glinthawk::gpu;

CUDAInfo::CUDAInfo()
{
  hipDeviceProp_t prop;
  int device;

  hipGetDevice( &device );
  hipGetDeviceProperties( &prop, device );

  max_threads_per_block_ = prop.maxThreadsPerBlock;
}
