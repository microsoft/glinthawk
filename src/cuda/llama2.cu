#include "hip/hip_runtime.h"
#include "llama2.cuh"

#include <cmath>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <source_location>

#include <fcntl.h>
#include <glog/logging.h>
#include <sys/stat.h>

#include <hipblas.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>

#include "util/exception.hh"
#include "util/file_descriptor.hh"
#include "util/ring_buffer.hh"

using namespace std;
using namespace glinthawk::gpu;

void CHECK_CUDA( const hipError_t err, const source_location location = source_location::current() )
{
  if ( err != hipSuccess ) {
    throw runtime_error( "CUDA error: " + string( hipGetErrorString( err ) ) + " (" + location.file_name() + ":"
                         + std::to_string( location.line() ) + ")" );
  }
}

namespace {

static hipblasHandle_t handle;

}

namespace ops {

__global__ void normalize_and_scale( float* output,
                                     const float* x,
                                     const float* weight,
                                     const int size,
                                     const float ss )
{
  const int i = threadIdx.x;
  output[i] = weight[i] * ss * x[i];
}

void rmsnorm( float* output, const float* x, const float* weight, const int size )
{
  // calculate sum of squares
  float ss = 0.0f;

  hipblasSdot( handle, size, x, 1, x, 1, &ss );
  ss /= size;
  ss += 1e-5f;
  ss = 1.0f / sqrtf( ss );

  normalize_and_scale<<<1, size>>>( output, x, weight, size, ss );
}

void softmax( float* _x, const int size )
{
  thrust::device_ptr<float> x { _x };

  const float max_val = *thrust::max_element( x, x + size );
  const float sum = thrust::transform_reduce(
    x, x + size, [max_val] __device__( const float x ) { return expf( x - max_val ); }, 0.0f, thrust::plus<float>() );
  thrust::transform( x, x + size, x, [sum] __device__( const float x ) { return x / sum; } );
}

void sample( const float* probabilities, const int n, int* output ) { throw runtime_error( "not implemented" ); }

void argmax( const float* _v, const int n, int* _output )
{
  thrust::device_ptr<const float> v { _v };
  thrust::device_ptr<int> output { _output };

  const auto it = thrust::max_element( v, v + n );
  *output = thrust::distance( v, it );
}

void accum( float* a, const float* b, const int size )
{
  float alpha = 1.0f;
  hipblasSaxpy( handle, size, &alpha, b, 1, a, 1 );
}

// void rmsnorm( float* o, const float* x, const float* weight, const int size );
// void softmax( float* x, const int size );

void matmul( float* xout, const float* x, const float* W, const int n, const int d )
{
  float alpha = 1.0f;
  float beta = 0.0f;

  // W(d,n) @ x(n,) -> xout(d,)
  hipblasSgemv( handle, HIPBLAS_OP_T, n, d, &alpha, W, n, x, 1, &beta, xout, 1 );
}

void silu( float* _hb, float* _hb2, const int hidden_dim )
{
  thrust::device_ptr<float> hb { _hb };
  thrust::device_ptr<float> hb2 { _hb2 };

  thrust::transform(
    hb, hb + hidden_dim, hb, [] __device__( float x ) { return x * ( 1.0f / ( 1.0f + expf( -x ) ) ); } );
  thrust::transform( hb, hb + hidden_dim, hb2, hb, thrust::multiplies<float>() );
}

// int sample( const float* probabilities, const int n );
// int argmax( const float* v, const int n );

}

Llama2::Config::Config( const filesystem::path& weights_path )
{
  ifstream fin { weights_path, ios::binary };
  CHECK( fin ) << "Failed to open weights file: " << weights_path;

  fin.read( reinterpret_cast<char*>( this ), sizeof( *this ) );

  vocab_size = abs( vocab_size );

  CHECK_GT( dim, 0 ) << "Transformer dimension must be positive.";
  CHECK_GT( hidden_dim, 0 ) << "FFN hidden dimension must be positive.";
  CHECK_GT( n_layers, 0 ) << "Number of layers must be positive.";
  CHECK_GT( n_heads, 0 ) << "Number of query heads must be positive.";
  CHECK_GT( n_kv_heads, 0 ) << "Number of key/value heads must be positive.";
  CHECK_GT( vocab_size, 0 ) << "Vocabulary size must be positive.";
  CHECK_GT( seq_len, 0 ) << "Sequence length must be positive.";

  LOG( INFO ) << "Loaded config: " << to_string();
}

string Llama2::Config::to_string() const
{
  ostringstream oss;
  oss << "{ ";
  oss << "dim: " << dim << ", ";
  oss << "hidden_dim: " << hidden_dim << ", ";
  oss << "n_layers: " << n_layers << ", ";
  oss << "n_heads: " << n_heads << ", ";
  oss << "n_kv_heads: " << n_kv_heads << ", ";
  oss << "vocab_size: " << vocab_size << ", ";
  oss << "seq_len: " << seq_len;
  oss << " }";
  return oss.str();
}

Llama2::Vocabulary::Vocabulary( const Config& config, const std::filesystem::path& vocabulary_path )
{
  ifstream fin { vocabulary_path, ios::binary };
  int len = 0;

  for ( int i = 0; i < config.vocab_size; i++ ) {
    CHECK( fin.read( reinterpret_cast<char*>( &len ), sizeof( int ) ) ) << "Failed to read vocabulary entry length.";
    CHECK_GT( len, 0 ) << "Vocabulary entry length must be positive.";

    string val;
    val.resize( len );
    CHECK( fin.read( val.data(), val.length() ) ) << "Failed to read vocabulary entry.";

    token_to_word_.push_back( val );
    word_to_token_.emplace( val, i );
  }

  LOG( INFO ) << "Loaded vocabulary of size " << config.vocab_size << " from " << vocabulary_path;
}

string Llama2::Vocabulary::get_word( int token ) const
{
  CHECK_GE( token, 0 ) << "Token index must be non-negative.";
  CHECK_LT( token, token_to_word_.size() ) << "Token index out of bounds.";
  return token_to_word_[token];
}

int Llama2::Vocabulary::get_token( const string& word ) const
{
  auto it = word_to_token_.find( word );
  CHECK( it != word_to_token_.end() ) << "Unknown word: " << word;
  return it->second;
}

Llama2::BaseWeights::BaseWeights( const Config& config, const float* model )
{
  auto ptr = model;
  this->token_embedding_table = ptr;

  // skip over all the layer weights
  ptr += config.vocab_size * config.dim
         + config.n_layers * ( 2 * config.dim + 4 * config.dim * config.dim + 3 * config.dim * config.hidden_dim );

  const int head_size = config.dim / config.n_heads;

  this->rms_final_weight = ptr;
  this->freq_cis_real = ( ptr += config.dim );
  this->freq_cis_imag = ( ptr += config.seq_len * head_size / 2 );

  // TODO shared_weights is assumed to be true, fix
  // wcls = true ? token_embedding_table : ( ptr += config.seq_len * head_size / 2 );
  this->wcls = token_embedding_table;
}

Llama2::LayerWeights::LayerWeights( const Config& config, const float* model, const int layer_num )
{
  auto ptr = model;

  // base pointers
  auto base_rms_att_weight = ( ptr += config.vocab_size * config.dim );
  auto base_wq = ( ptr += config.n_layers * config.dim );
  auto base_wk = ( ptr += config.n_layers * config.dim * config.dim );
  auto base_wv = ( ptr += config.n_layers * config.dim * config.dim );
  auto base_wo = ( ptr += config.n_layers * config.dim * config.dim );
  auto base_rms_ffn_weight = ( ptr += config.n_layers * config.dim * config.dim );
  auto base_w1 = ( ptr += config.n_layers * config.dim );
  auto base_w2 = ( ptr += config.n_layers * config.dim * config.hidden_dim );
  auto base_w3 = ( ptr += config.n_layers * config.hidden_dim * config.dim );

  this->rms_att_weight = base_rms_att_weight + layer_num * config.dim;
  this->rms_ffn_weight = base_rms_ffn_weight + layer_num * config.dim;
  this->wq = base_wq + layer_num * config.dim * config.dim;
  this->wk = base_wk + layer_num * config.dim * config.dim;
  this->wv = base_wv + layer_num * config.dim * config.dim;
  this->wo = base_wo + layer_num * config.dim * config.dim;
  this->w1 = base_w1 + layer_num * config.dim * config.hidden_dim;
  this->w2 = base_w2 + layer_num * config.hidden_dim * config.dim;
  this->w3 = base_w3 + layer_num * config.hidden_dim * config.dim;
}

Llama2::Llama2( const std::filesystem::path& tokenizer_path,
                const filesystem::path& model_path,
                const int32_t start_layer,
                const int32_t end_layer )
  : model_ptr_( [&] {
    const auto model_size = filesystem::file_size( model_path );
    FileDescriptor model_fd { CHECK_SYSCALL( "open", open( model_path.c_str(), O_RDONLY ) ) };
    MMap_Region model_mmap { nullptr, model_size, PROT_READ, MAP_PRIVATE, model_fd.fd_num(), 0 };
    void* ptr;

    CHECK_CUDA( hipMalloc( &ptr, model_size ) );
    CHECK_CUDA( hipMemcpy( ptr, model_mmap.addr(), model_size, hipMemcpyHostToDevice ) );

    return reinterpret_cast<const float*>( ptr ) + sizeof( Config ) / sizeof( float );
  }() )
  , config_( model_path )
  , start_layer_num_( start_layer )
  , end_layer_num_( end_layer == -1 ? config_.n_layers - 1 : end_layer )
  , base_weights_( config_, model_ptr_ )
  , layer_weights_( [&] {
    CHECK_GE( start_layer_num_, 0 ) << "Start layer must be non-negative.";
    CHECK_LT( end_layer_num_, config_.n_layers ) << "End layer must be less than the number of layers.";

    vector<LayerWeights> layers( config_.n_layers );
    for ( int i = start_layer_num_; i <= end_layer_num_; i++ ) {
      new ( &layers[i] ) LayerWeights { config_, model_ptr_, i };
    }

    return layers;
  }() )
  , vocabulary_( config_, tokenizer_path )
  , state_( config_, start_layer_num_, end_layer_num_ )
{
  hipblasCreate( &handle );
}

Llama2::RunState::RunState( const Config& config, const int32_t start_layer, const int32_t end_layer )
  : buffer_( [&] {
    void* ptr;
    const auto size = sizeof( float )
                      * ( config.dim * 5 + config.hidden_dim * 2 + config.n_heads * config.seq_len + config.vocab_size
                          + config.n_heads );

    CHECK_CUDA( hipMalloc( &ptr, size ) );
    return reinterpret_cast<float*>( ptr );
  }() )
  , x( [&] {
    void* ptr;
    const auto size = sizeof( float ) * config.dim;
    CHECK_CUDA( hipMalloc( &ptr, size ) );
    return reinterpret_cast<float*>( ptr );
  }() )
  , xb( buffer_ )
  , xb2( xb + config.dim )
  , q( xb2 + config.dim )
  , k( q + config.dim )
  , v( k + config.dim )
  , hb( v + config.dim )
  , hb2( hb + config.hidden_dim )
  , att( hb2 + config.hidden_dim )
  , logits( att + config.n_heads * config.seq_len )
  , temp_softmax( logits + config.vocab_size )
  , kv_cache( config, start_layer, end_layer )
{
}

Llama2::RunState::KVCache::KVCache( const Config& config, const int32_t start_layer, const int32_t end_layer )
  : start_layer_( start_layer )
  , end_layer_( end_layer )
  , buffer_( [&] {
    void* ptr;
    const auto size = sizeof( float ) * config.seq_len * ( end_layer - start_layer + 1 ) * config.dim * 2;
    CHECK_CUDA( hipMalloc( &ptr, size ) );
    return reinterpret_cast<float*>( ptr );
  }() )
  , seq_len_( config.seq_len )
  , dim_( config.dim )
  , n_layers_( end_layer_ - start_layer_ + 1 )
  , head_size_( config.dim / config.n_heads )
{
}

float* Llama2::RunState::KVCache::key( int layer, const int step, const int head )
{
  layer -= start_layer_;
  return buffer_ + step * ( n_layers_ * dim_ * 2 ) + layer * ( dim_ * 2 ) + head * head_size_;
}

float* Llama2::RunState::KVCache::value( int layer, const int step, const int head )
{
  layer -= start_layer_;
  return buffer_ + step * ( n_layers_ * dim_ * 2 ) + layer * ( dim_ * 2 ) + head * head_size_ + dim_;
}

void Llama2::RunState::KVCache::pop() { throw runtime_error( "KVCache::pop() not implemented" ); }

void Llama2::pass_begin( const int token )
{
  // copy the token embedding into the state
  const float* content_row = base_weights_.token_embedding_table + token * config_.dim;
  CHECK_CUDA( hipMemcpy( state_.x, content_row, config_.dim * sizeof( float ), hipMemcpyDeviceToDevice ) );
}

__global__ void do_rope( const int head_size,
                         const int n_heads,
                         const float* freq_cis_real_row,
                         const float* freq_cis_imag_row,
                         float* state_q,
                         float* state_k )
{
  const int head_num = blockIdx.x;
  const int elem_idx = 2 * threadIdx.x;

  // apply RoPE rotation to the q and k vectors for each head
  // get the q and k vectors for this head
  float* q = state_q + head_num * head_size;
  float* k = state_k + head_num * head_size;

  // rotate q and k by the freq_cis_real and freq_cis_imag
  const float q0 = q[elem_idx];
  const float q1 = q[elem_idx + 1];
  const float k0 = k[elem_idx];
  const float k1 = k[elem_idx + 1];
  const float fcr = freq_cis_real_row[elem_idx / 2];
  const float fci = freq_cis_imag_row[elem_idx / 2];
  q[elem_idx] = q0 * fcr - q1 * fci;
  q[elem_idx + 1] = q0 * fci + q1 * fcr;
  k[elem_idx] = k0 * fcr - k1 * fci;
  k[elem_idx + 1] = k0 * fci + k1 * fcr;
}

__global__ void attention_0( const float* all_q,
                             const float* kv_cache,
                             float* att,
                             const int layer_num,
                             const int n_layers,
                             const int seq_len,
                             const int head_size,
                             const int dim )
{
  const int head_num = threadIdx.x;
  const int token_pos = blockIdx.x;

  att += head_num * seq_len;
  const float* q = all_q + head_num * head_size;
  const float* k = kv_cache + token_pos * ( n_layers * dim * 2 ) + layer_num * ( dim * 2 ) + head_num * head_size;

  float score = 0.0f;
  for ( int i = 0; i < head_size; i++ ) {
    score += q[i] * k[i];
  }
  score /= sqrtf( head_size );

  // save the score to the attention buffer
  att[token_pos] = score;
}

__global__ void find_max_for_rows( const float* att,
                                   float* output,
                                   const int token_pos,
                                   const int n_heads,
                                   const int seq_len )
{
  const int head_num = threadIdx.x;
  att += head_num * seq_len;

  float max_value = att[0];
  for ( int i = 1; i <= token_pos; i++ ) {
    max_value = max( max_value, att[i] );
  }

  output[head_num] = max_value;
}

__global__ void subtract_and_expf( const float* values, float* att, const int n_heads, const int seq_len )
{
  const int head_num = threadIdx.x;
  const int token_pos = blockIdx.x;

  att += head_num * seq_len;
  att[token_pos] = expf( att[token_pos] - values[head_num] );
}

__global__ void sum_rows( float* att, float* output, const int token_pos, const int n_heads, const int seq_len )
{
  const int head_num = threadIdx.x;
  att += head_num * seq_len;

  float sum = 0.0;
  for ( int i = 0; i <= token_pos; i++ ) {
    sum += att[i];
  }

  output[head_num] = sum;
}

__global__ void normalize_by_sum( float* att, const float* sums, const int n_heads, const int seq_len )
{
  const int head_num = threadIdx.x;
  const int token_pos = blockIdx.x;

  att += head_num * seq_len;
  att[token_pos] /= sums[head_num];
}

void attention_softmax( float* att, const int token_pos, const int seq_len, const int n_heads, float* temp_buffer )
{
  float* head_values = temp_buffer;

  // (1) find the max value for each head (each row)
  find_max_for_rows<<<1, n_heads>>>( att, head_values, token_pos, n_heads, seq_len );

  // (2) exp(att - max)
  subtract_and_expf<<<token_pos + 1, n_heads>>>( head_values, att, n_heads, seq_len );

  // (3) sum each row
  sum_rows<<<1, n_heads>>>( att, head_values, token_pos, n_heads, seq_len );

  // (4) normalize each row by its sum
  normalize_by_sum<<<token_pos + 1, n_heads>>>( att, head_values, n_heads, seq_len );
}

__global__ void attention_2( float* att,
                             const float* kv_cache,
                             float* xb,
                             const int layer_num,
                             const int n_layers,
                             const int seq_len,
                             const int head_size,
                             const int dim )
{
  const int head_num = threadIdx.x;
  const int token_pos = blockIdx.x;

  att += head_num * seq_len;
  xb += head_num * head_size;

  const float a = att[token_pos];
  const float* v = kv_cache + token_pos * ( n_layers * dim * 2 ) + layer_num * ( dim * 2 ) + head_num * head_size + dim;

  for ( int i = 0; i < head_size; i++ ) {
    atomicAdd( &xb[i], a * v[i] );
  }
}

void Llama2::transformer_layer( const int32_t layer_num, const int token_pos )
{
  float* const x = state_.x;
  const int dim = config_.dim;
  const int hidden_dim = config_.hidden_dim;
  const int head_size = dim / config_.n_heads;

  // pluck out the "pos" row of freq_cis_real and freq_cis_imag
  const float* freq_cis_real_row = base_weights_.freq_cis_real + token_pos * head_size / 2;
  const float* freq_cis_imag_row = base_weights_.freq_cis_imag + token_pos * head_size / 2;

  const auto& layer_weights = layer_weights_[layer_num];

  // attention rmsnorm
  ops::rmsnorm( state_.xb, x, layer_weights.rms_att_weight, dim );

  // qkv matmuls for this position
  ops::matmul( state_.q, state_.xb, layer_weights.wq, dim, dim );
  ops::matmul( state_.k, state_.xb, layer_weights.wk, dim, dim );
  ops::matmul( state_.v, state_.xb, layer_weights.wv, dim, dim );

  do_rope<<<config_.n_heads, head_size / 2>>>(
    head_size, config_.n_heads, freq_cis_real_row, freq_cis_imag_row, state_.q, state_.k );

  float* k_cache_pos = state_.kv_cache.key( layer_num, token_pos );
  float* v_cache_pos = state_.kv_cache.value( layer_num, token_pos );

  // save key,value at this time step (pos) to our kv cache
  CHECK_CUDA( hipMemcpy( k_cache_pos, state_.k, dim * sizeof( float ), hipMemcpyDeviceToDevice ) );
  CHECK_CUDA( hipMemcpy( v_cache_pos, state_.v, dim * sizeof( float ), hipMemcpyDeviceToDevice ) );

  // multihead attention. for each head and for each token up to and including the current one
  attention_0<<<token_pos + 1, config_.n_heads>>>(
    state_.q, state_.kv_cache.buffer_, state_.att, layer_num, config_.n_layers, config_.seq_len, head_size, dim );

  // softmax
  attention_softmax( state_.att, token_pos, config_.seq_len, config_.n_heads, state_.temp_softmax );

  CHECK_CUDA( hipMemset( state_.xb, 0, dim * sizeof( float ) ) );

  attention_2<<<token_pos + 1, config_.n_heads>>>(
    state_.att, state_.kv_cache.buffer_, state_.xb, layer_num, config_.n_layers, config_.seq_len, head_size, dim );
  // end of multihead attention

  // final matmul to get the output of the attention
  ops::matmul( state_.xb2, state_.xb, layer_weights.wo, dim, dim );

  // residual connection back into x
  ops::accum( x, state_.xb2, dim );

  // ffn rmsnorm
  ops::rmsnorm( state_.xb, x, layer_weights.rms_ffn_weight, dim );

  // now for ffn in we have: self.w2(F.silu(self.w1(x)) * self.w3(x))
  // first calculate self.w1(x) and self.w3(x)
  ops::matmul( state_.hb, state_.xb, layer_weights.w1, dim, hidden_dim );
  ops::matmul( state_.hb2, state_.xb, layer_weights.w3, dim, hidden_dim );

  ops::silu( state_.hb, state_.hb2, hidden_dim );

  // final matmul to get the output of the ffn
  ops::matmul( state_.xb, state_.hb, layer_weights.w2, hidden_dim, dim );

  // residual connection
  ops::accum( x, state_.xb, dim );
}

void Llama2::pass_end()
{
  float* x = state_.x;

  // final rmsnorm
  ops::rmsnorm( x, x, base_weights_.rms_final_weight, config_.dim );

  // classifier into logits
  ops::matmul( state_.logits, x, base_weights_.wcls, config_.dim, config_.vocab_size );
}

pair<int, string> Llama2::forward( const int token )
{
  if ( token_pos >= config_.seq_len ) {
    return { 2, {} };
  }

  pass_begin( token );

  for ( int layer_num = start_layer_num_; layer_num <= end_layer_num_; layer_num++ ) {
    transformer_layer( layer_num, token_pos );
  }

  pass_end();

  int next_token;
  int* next_token_device;
  CHECK_CUDA( hipMalloc( &next_token_device, sizeof( int ) ) );

  if ( temperature_ == 0.0f ) {
    // greedy argmax sampling
    ops::argmax( state_.logits, config_.vocab_size, next_token_device );
  } else {
    // apply the temperature to the logits
    for ( int q = 0; q < config_.vocab_size; q++ ) {
      state_.logits[q] /= temperature_;
    }

    // apply softmax to the logits to get the probabilities for next token
    ops::softmax( state_.logits, config_.vocab_size );

    // we now want to sample from this distribution to get the next token
    ops::sample( state_.logits, config_.vocab_size, next_token_device );
  }

  token_pos++;
  CHECK_CUDA( hipMemcpy( &next_token, next_token_device, sizeof( int ), hipMemcpyDeviceToHost ) );
  return { next_token, vocabulary_.get_word( next_token ) };
}
